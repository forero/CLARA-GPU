#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <string.h>

#include "io.h"
#include "struct.h"
/*this makes the compiler happy*/
#include "vector.cu"


/*
  RNG kernel initialization
*/
__global__ void initRNG(hiprandState *const rngStates,
                        const unsigned int seed)
{
    // Determine thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialise the RNG
    hiprand_init(seed, tid, 0, &rngStates[tid]);
}

/*
  FLOAT random number generator
*/
__device__ void getPoint(FLOAT *x, hiprandState *state)
{
  *x = hiprand_normal_double(state);
}


__device__ void RND_spherical(FLOAT *vec, hiprandState *state)
/*vector randomly distributed over the sphere*/
{
    FLOAT theta, phi;
    theta = acos(2.0*(hiprand_uniform(state) -0.5));
    phi = 2.0*PI*hiprand_uniform(state);
    vec[0] = sin(theta)*cos(phi);
    vec[1] = sin(theta)*sin(phi);
    vec[2] = cos(theta);
}

__device__ void RND_lyman_parallel_vel(FLOAT *u_parallel, FLOAT x, FLOAT a, hiprandState *state, int *status) 
/* 
   Generates the parallel velocity to the photon propagation.
   First generates a random number \theta between -pi/2 and pi/2, then 
   it generates u_parallel through u_parallel = a \tan\theta + x
   then this value of u_parallel is kept if another random number 
   between [0,1] is smaller than \exp^{-u_parallel^2}. 
   Finally, the value of u_parallel is multiplied by the sign of x.       
*/
{
    int finished = 0;
    FLOAT tmp0, tmp1, tmp2;        
    int counter = 0;
    
    while (finished == 0) {
      tmp0 = (hiprand_uniform(state) - 0.5)*PI ;
      tmp1 = (a*tan(tmp0)) + fabs(x); 
      tmp2  = hiprand_uniform(state);
      if(tmp2 <= (exp(-(tmp1*tmp1)))) finished = 1;		
      counter++;		
      if(counter > MAX_VEL_ITER) {
	finished = 1;		    
	*status = EXCEEDED_ITERATIONS;
      }	
    }

    if(x > 0.0){
      *u_parallel = tmp1;
    }else{    
      *u_parallel = -tmp1;
    }
}

__device__ void RND_lyman_perp_vel(FLOAT *u_1, FLOAT *u_2, hiprandState *state)
/* 
   Genereates magnitudes for the atom's perpendicular velocities using
   Box&Muller method. 
*/
{
  FLOAT tmp1, tmp2;
  FLOAT vel_1, vel_2;
  tmp1 = hiprand_uniform(state);
  tmp2 = hiprand_uniform(state);
  vel_1 = sqrt(-log(tmp1))*cos(2.0*PI*tmp2);
  vel_2 = sqrt(-log(tmp1))*sin(2.0*PI*tmp2);
  *u_1 = vel_1;
  *u_2 = vel_2;  
  return;
}


__device__ void RND_pair(FLOAT *r_1, FLOAT *r_2, hiprandState *state){
  /*generates a pair of random numbers with norm less than one*/
  int finished;
  FLOAT rand_1, rand_2;

  finished = 0;
  while (finished == 0) {
    rand_1 = 2.0*(hiprand_uniform(state) - 0.5);
    rand_2 = 2.0*(hiprand_uniform(state) - 0.5);
    if(rand_1*rand_1 + rand_2*rand_2 < 1.0){
      finished = 1;
    }
  }    
  *r_1 = rand_1;
  *r_2 = rand_2;
}


__device__ void RND_lyman_atom(FLOAT *Vel, FLOAT *DirPhoton, FLOAT *DirOutPhoton, FLOAT x, FLOAT a ,hiprandState *state, int *status)
/* Obtains a random velocity for the hydrogen atom
   the velocity is in units of the termal velocity.*/
{
    int i;
    FLOAT LocalVel[3];
    FLOAT x_axis[3];
    FLOAT y_axis[3];
    FLOAT z_axis[3];
    FLOAT rand_axis[3];
    FLOAT R_1, R_2, R_3, T, mu, iso;
    FLOAT  x_corewing;
    FLOAT exponent;

    /*get first the parallel velocity*/
    RND_lyman_parallel_vel(&(LocalVel[2]), x, a, state, status);

    /*get the perpendicular velocity*/
    RND_lyman_perp_vel(&(LocalVel[0]), &(LocalVel[1]), state);

    /*get the axis in the coordinate system of the atom, where 
      the z direction is the propagation direction of the photon*/
    for(i=0;i<3;i++){
	z_axis[i] = DirPhoton[i];
    }

    /*get another random vector*/
    RND_spherical(rand_axis, state);

    /*make the cross product and get y_axis*/
    cross_product(z_axis, rand_axis, y_axis);

    /*make the cross product and get x_axis*/
    cross_product(y_axis, z_axis, x_axis);

    /*normalize the vectors*/
    normalize(x_axis);
    normalize(y_axis);
    normalize(z_axis);

    /*see if they are perpendicular*/
    rand_axis[0] = point_product(x_axis, z_axis);
    rand_axis[1] = point_product(x_axis, y_axis);
    rand_axis[2] = point_product(y_axis, z_axis);
    if(fabs(rand_axis[0]) + fabs(rand_axis[1])+ fabs(rand_axis[2])>1.0e-10){
      *status = NULL_NORM;
    }

    /*Now make the transformation into the coordinate frame of the lab*/
    for(i=0;i<3;i++){
	Vel[i] = LocalVel[0]*x_axis[i] + LocalVel[1]*y_axis[i] + LocalVel[2]*z_axis[i];
    }

    /*
      now get the outgoing direction of the photon, 
      taking advantage of the vector basis I have just generated
      Here I just take the value of dijkstra for the wing.
    */

    /*first define if it's in the core or not*/    

    x_corewing = 1.59 - 0.60*log10(a) - 0.03*log10(a)*log10(a);

    R_1 = hiprand_uniform(state);

    iso = hiprand_uniform(state);
    if(iso<(1.0/3.0)){/*isotropic*/
      mu = (2.0*R_1 - 1.0);
    }else{
      if(fabs(x)<x_corewing){			/*In the core*/
	/*now we make the decision if it's isotropic or not*/
	T = (1.0/7.0)*(14.0  - 24.0*R_1  + sqrt(245.0 - 672.0*R_1 + 576*R_1*R_1));	  
	exponent = 1.0/3.0; /*this makes cuda compiler happy*/
	mu = 1.0/(pow(T,exponent)) - pow(T, exponent);	
      }else{
	T = 2.0 - 4.0*R_1  + sqrt(5.0 -16.0*R_1 + 16*R_1*R_1);
	exponent = 1.0/3.0; /*this makes cuda compiler happy*/
	mu = 1.0/(pow(T,exponent)) - pow(T, exponent);	
      }
    }
    
    RND_pair(&R_1, &R_2, state);    
    R_3 = R_1*R_1 + R_2*R_2;
    for(i=0;i<3;i++){
	DirOutPhoton[i] = 
	    sqrt((1.0-(mu*mu))/R_3)*R_1*x_axis[i] + 
	    sqrt((1.0-(mu*mu))/R_3)*R_2*y_axis[i] + 
	    mu*z_axis[i];
    }
}


/*This makes the compiler happy*/
#include "propagate.cu"
#include "test.cu"

/*
  This is CLARA's core.
  This routine computes all the scatterings until the photon escapes.
*/
__global__ void scatterStep(FLOAT *x, FLOAT *p, FLOAT *k, int * n_scatter, int *status_ID, int N, hiprandState *const rngStates, setup *S)
{
  /*physical variables that define the photon state*/
  FLOAT pos[3];
  FLOAT dir[3];
  FLOAT r_travel, x_photon, x_comoving;
  FLOAT last_x;

  /*physical variables that define the medium*/
  FLOAT a, nu_doppler, n_HI, v_thermal, BulkVel[3], temperature;  

  /*status variables*/
  int photon_status;
  int program_status;
  int n_iter=0;
  int i;

  /*memory thread on the GPU*/
  int id = blockIdx.x*blockDim.x + threadIdx.x;
  int idx = blockIdx.x*blockDim.x + (threadIdx.x*3  + 0); 
  int idy = blockIdx.x*blockDim.x + (threadIdx.x*3  + 1);
  int idz = blockIdx.x*blockDim.x + (threadIdx.x*3  + 2);


  FLOAT f=0.0;
  FLOAT px=0.0;
  FLOAT py=0.0;
  FLOAT pz=0.0;
  FLOAT norm;


  /*Initializes the random number generator*/
  hiprandState localState = rngStates[id];

  /*Make the initialization for the photon*/
  pos[0] = p[idx];
  pos[1] = p[idy];
  pos[2] = p[idz];
  dir[0] = k[idx];
  dir[1] = k[idy];
  dir[2] = k[idz];
  x_photon = x[id];
  photon_status = status_ID[id]; 
  
  if (id < N){    
    while(PropagateIsInside(pos[0], pos[1], pos[2], S) && (photon_status==ACTIVE) && (n_iter<MAX_ITER)){

      /* get the temperature at this point*/
      PropagateGetTemperature(&temperature, &(pos[0]), S);
      
      /* get the number density at this point*/
      PropagateGetNumberDensity(&n_HI, pos, S);
      
      /*get the bulk velocity of the fluid at this point*/
      PropagateGetBulkVel(BulkVel, pos, S);
      
      /*Get the thermal velocity and doppler broadening*/
      nu_doppler = CONSTANT_NU_DOPPLER*sqrt(temperature/10000.0); /* in cm/s */
      a = Lya_nu_line_width_CGS/(2.0*nu_doppler);
      v_thermal = (nu_doppler/Lya_nu_center_CGS)*C_LIGHT;/*In cm/s*/
      
      /*change the value of the frequency to one comoving with the fluid*/
      PropagateLorentzFreqChange(&x_photon, dir, BulkVel, v_thermal, -1); 
            
      /*change the direction of the photon to the fluid frame*/
      PropagateLorentzDirChange(&(dir[0]), BulkVel, -1);
                  
      /*--------------------------------------------------------------------------*/
      /*Change the frequency and the Propagation direction, find the displacement*/	
      photon_status = PropagateStep(&x_photon, &(dir[0]), &r_travel, &a, &localState, S);	    	
      /*--------------------------------------------------------------------------*/
            
      /*Change the new direction to the lab frame value*/
      PropagateLorentzDirChange(&(dir[0]), BulkVel, 1);
      
      /*Change the frequency comoving to the lab frame value*/
      PropagateLorentzFreqChange(&x_photon, dir, BulkVel, v_thermal, 1); 
      
      /*Update the position*/
      for(i=0;i<3;i++){
	pos[i] += r_travel*dir[i];	    
      }
      
      n_iter++;
    }
    __syncthreads();
  }

  /*update the photon status*/
  if(photon_status==ACTIVE){
    photon_status = OUT_OF_BOX;
  }
  if(n_iter>= MAX_ITER){
    photon_status = SATURATED_ITERATIONS;
  }


  /*update the values*/
  p[idx] = pos[0];
  p[idy] = pos[1];
  p[idz] = pos[2];
  k[idx] = dir[0];
  k[idy] = dir[1];
  k[idz] = dir[2];
  x[id] = x_photon;  
  status_ID[id] = photon_status;
  n_scatter[id] = n_iter;

  __syncthreads();
}


/*
  This is the main driver for CLARA.
  Initializes the memory on the device for:
  - Photons positions.
  - Photons direction of propagation.
  - Photons frequency.
  - Global setup (densities, velocities, temperatures)

  This is also the place where the main GPU characteristics have to be setup:
  - Number of blocks
  - Number of threads
*/
extern "C" void scatter_bunch(FLOAT *x, FLOAT *p, FLOAT *k, int *n_scatter, int *status_ID, int min_id, int max_id){
  FLOAT *x_aux;
  FLOAT *p_aux;
  FLOAT *k_aux;
  int *n_scatter_aux;
  int *status_ID_aux;
  FLOAT *x_aux_d;
  FLOAT *p_aux_d;
  FLOAT *k_aux_d;
  int *n_scatter_aux_d;
  int *status_ID_aux_d;

  int n_aux;
  int blockSize;
  int nBlocks;
  int i,l;
  setup *S;
  unsigned int m_seed;
  unsigned int m_numSims;
  unsigned int m_device;
  unsigned int m_threadBlockSize;  
  struct hipDeviceProp_t     deviceProperties;
  hipError_t cudaResult = hipSuccess;

  hipSetDevice(0);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);

 char msg[256];
        sprintf(msg, "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
                (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
        printf("%s", msg);

  cudaResult = hipGetLastError();
  if (cudaResult!=hipSuccess){
    printf( "Error after device!\n" );
    printf("CUDA error: %s\n", hipGetErrorString(cudaResult));
  }  


  
  m_device = 0;
  // Get device properties
  cudaResult = hipGetDeviceProperties(&deviceProperties, m_device);
  fprintf(stdout, "Device Properties:\n Multiproc count %d\n", deviceProperties.multiProcessorCount );

  //allocate and copy the global setup structure 
  hipMalloc((void**) &S, sizeof(setup));
  hipMemcpy(S, &All, sizeof(setup), hipMemcpyHostToDevice);

  //allocate auxiliary variables  
  n_aux = max_id - min_id;
  if(!(x_aux = (FLOAT *)malloc(n_aux * sizeof(FLOAT)))){
    fprintf(stderr, "Problem allocating the auxiliary array\n");
    exit(1);
  }

  if(!(p_aux = (FLOAT *)malloc(3 * n_aux * sizeof(FLOAT)))){
    fprintf(stderr, "Problem allocating the auxiliary array\n");
    exit(1);
  }

  if(!(k_aux = (FLOAT *)malloc(3 * n_aux * sizeof(FLOAT)))){
    fprintf(stderr, "Problem allocating the auxiliary array\n");
    exit(1);
  }

  if(!(n_scatter_aux = (int *)malloc(n_aux * sizeof(int)))){
    fprintf(stderr, "Problem allocating the auxiliary array\n");
    exit(1);
  }

  if(!(status_ID_aux = (int *)malloc(n_aux * sizeof(int)))){
    fprintf(stderr, "Problem allocating the auxiliary array\n");
    exit(1);
  }

  //fill the auxiliary variables
  for(i=0;i<n_aux;i++){
    x_aux[i] = x[min_id+i];
    for(l=0;l<3;l++){
      p_aux[3*i + l] = p[3*(min_id+i)+l];
      k_aux[3*i + l] = k[3*(min_id+i)+l];
    }
    n_scatter_aux[i] = n_scatter[min_id + i];
    status_ID_aux[i] = status_ID[min_id + i];
  }


  // allocate memory on device
  hipMalloc((void **) &x_aux_d, n_aux * sizeof(FLOAT));
  hipMalloc((void **) &p_aux_d, 3 * n_aux * sizeof(FLOAT));
  hipMalloc((void **) &k_aux_d, 3 * n_aux * sizeof(FLOAT));
  hipMalloc((void **) &n_scatter_aux_d, n_aux * sizeof(int));
  hipMalloc((void **) &status_ID_aux_d, n_aux * sizeof(int));

  // copy data from host to device
  hipMemcpy(x_aux_d, x_aux, sizeof(FLOAT) * n_aux, hipMemcpyHostToDevice);
  hipMemcpy(p_aux_d, p_aux, 3 * sizeof(FLOAT) * n_aux, hipMemcpyHostToDevice);
  hipMemcpy(k_aux_d, k_aux, 3 * sizeof(FLOAT) * n_aux, hipMemcpyHostToDevice);
  hipMemcpy(n_scatter_aux_d, n_scatter_aux, sizeof(int) * n_aux, hipMemcpyHostToDevice);
  hipMemcpy(status_ID_aux_d, status_ID_aux, sizeof(int) * n_aux, hipMemcpyHostToDevice);



  blockSize = 512; // This is the number of threads inside a block
  nBlocks = (3*n_aux)/blockSize + (n_aux%blockSize == 0?0:1); // This is the number of blocks
  fprintf(stdout, "nBlocks %d\n", nBlocks);

  //allocate memory for RNG states
  hiprandState *d_rngStates = 0;
  cudaResult = hipMalloc((void **)&d_rngStates, blockSize * nBlocks * sizeof(hiprandState));

  // Initialise RNG
  m_seed = min_id;
  initRNG<<<nBlocks, blockSize>>>(d_rngStates, m_seed);

  // Make the random step until all the photons escape
  scatterStep <<< nBlocks, blockSize >>> (x_aux_d, p_aux_d, k_aux_d, n_scatter_aux_d, status_ID_aux_d, n_aux, d_rngStates, S);

  // copy data from device to host
  hipMemcpy(x_aux, x_aux_d, sizeof(FLOAT) * n_aux, hipMemcpyDeviceToHost);
  hipMemcpy(p_aux, p_aux_d, 3 * sizeof(FLOAT) * n_aux, hipMemcpyDeviceToHost);
  hipMemcpy(k_aux, k_aux_d, 3 * sizeof(FLOAT) * n_aux, hipMemcpyDeviceToHost);
  hipMemcpy(n_scatter_aux, n_scatter_aux_d, sizeof(int) * n_aux, hipMemcpyDeviceToHost);
  hipMemcpy(status_ID_aux, status_ID_aux_d, sizeof(int) * n_aux, hipMemcpyDeviceToHost);

  printf("%f\n", All.Tau);

  for(i=0;i<n_aux;i++){
    x[min_id+i] = x_aux[i];
    for(l=0;l<3;l++){
      p[3*(min_id+i)+l] = p_aux[3*i + l]; 
      k[3*(min_id+i)+l] = k_aux[3*i + l]; 
    }    
    n_scatter[min_id + i] = n_scatter_aux[i];
    status_ID[min_id + i] = status_ID_aux[i];
  }
}

