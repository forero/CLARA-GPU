#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#include <hiprand/hiprand_kernel.h>



// RNG init kernel
__global__ void initRNG(hiprandState *const rngStates,
                        const unsigned int seed)
{
    // Determine thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialise the RNG
    hiprand_init(seed, tid, 0, &rngStates[tid]);
}

__device__ void getPoint(float *x, hiprandState *state)
{
  *x = hiprand_normal_double(state);
}


__global__ void randomStep(float *a, int N, hiprandState *const rngStates)
{

  int idx = blockIdx.x*blockDim.x + threadIdx.x;

  // Initialise the RNG
  hiprandState localState = rngStates[idx];
  float x=0.0;

  if (idx<N){    
    getPoint(&x, &localState);
    while(fabs(x)<1.0){
      getPoint(&x, &localState);
    }
    a[idx] = x;
    //    while(fabs(a[idx])>3.0){
    //      a[idx] = ;
    //    }
  }
  
}

extern "C" void scatter_x(float *x, int min_id, int max_id){
  float *x_aux;
  float *x_aux_d;
  int n_aux;
  int blockSize;
  int nBlocks;
  int i;

  unsigned int m_seed;
  unsigned int m_numSims;
  unsigned int m_device;
  unsigned int m_threadBlockSize;  
  struct hipDeviceProp_t     deviceProperties;
  struct hipFuncAttributes funcAttributes;
  hipError_t cudaResult = hipSuccess;

  
  m_device = 0;
  // Get device properties
  cudaResult = hipGetDeviceProperties(&deviceProperties, m_device);

  fprintf(stdout, "Device Properties:\n Multiproc count %d\n", deviceProperties.multiProcessorCount );

  //allocate auxiliary variables  
  n_aux = max_id - min_id;

  if(!(x_aux = (float *)malloc(n_aux * sizeof(float)))){
    fprintf(stderr, "Problem allocating the auxiliary array\n");
    exit(1);
  }
  
  for(i=0;i<n_aux;i++){
    x_aux[i] = x[min_id+i];
  }

  // allocate memory on device
  hipMalloc((void **) &x_aux_d, n_aux * sizeof(float));

  // copy data from host to device
  hipMemcpy(x_aux_d, x_aux, sizeof(float) * n_aux, hipMemcpyHostToDevice);

  blockSize = 128; // This is the number of threads inside a block
  nBlocks = n_aux/blockSize + (n_aux%blockSize == 0?0:1); // This is the number of blocks

  //allocate memory for RNG states
  hiprandState *d_rngStates = 0;
  cudaResult = hipMalloc((void **)&d_rngStates, blockSize * nBlocks * sizeof(hiprandState));

  // Initialise RNG
  m_seed = min_id;
  initRNG<<<nBlocks, blockSize>>>(d_rngStates, m_seed);

  // Make the random step
  randomStep <<< nBlocks, blockSize >>> (x_aux_d, n_aux, d_rngStates);

  // copy data from device to host
  hipMemcpy(x_aux, x_aux_d, sizeof(float) * n_aux, hipMemcpyDeviceToHost);

  for(i=0;i<n_aux;i++){
    x[min_id+i] = x_aux[i];
  }
}


extern "C" void TransportPhotons(float *x, int n_photons){
  int pack_size, last_pack_size;  
  int n_packs;
  int i;

  pack_size = 400;
  
  n_packs = n_photons/pack_size;
  last_pack_size = n_photons%pack_size;
  
  fprintf(stdout, "Photons to transport %d\n", n_photons);
  fprintf(stdout, "%d packs of size %d\n", n_packs, pack_size);
  fprintf(stdout, "one last pack of size %d\n", last_pack_size);
  
  for(i=0;i<n_packs;i++){
    scatter_x(x, i*pack_size, (i+1)*pack_size);
  }  
  scatter_x(x, i*pack_size, i*pack_size + last_pack_size);  

}
