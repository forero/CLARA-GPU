#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include "scatter.h"
//#include "scatter_gpu.h"
//#include "scatter_cpu.h"
__global__ void randomStep(float *a, int N)
{

  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx<N) a[idx] = 3.14159;

}

void scatter_x(float *x, int min_id, int max_id){
  float *x_aux;
  float *x_aux_d;
  int n_aux;
  int blockSize;
  int nBlocks;
  int i;

  n_aux = max_id - min_id;

  if(!(x_aux = (float *)malloc(n_aux * sizeof(float)))){
    fprintf(stderr, "Problem allocating the auxiliary array\n");
    exit(1);
  }
  
  for(i=0;i<n_aux;i++){
    x_aux[i] = x[min_id+i];
  }

  // allocate memory on device
  hipMalloc((void **) &x_aux_d, n_aux * sizeof(float));

  // copy data from host to device
  hipMemcpy(x_aux_d, x_aux, sizeof(float) * n_aux, hipMemcpyHostToDevice);

  blockSize = 32; // This is the number of threads inside a block
  nBlocks = n_aux/blockSize + (n_aux%blockSize == 0?0:1); // This is the number of blocks

  // Make the random step
  randomStep <<< nBlocks, blockSize >>> (x_aux_d, n_aux);

  // copy data from device to host
  hipMemcpy(x_aux, x_aux_d, sizeof(float) * n_aux, hipMemcpyDeviceToHost);
}
