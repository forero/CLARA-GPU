#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "struct.h"

__device__ FLOAT point_product(FLOAT *vec_1, FLOAT *vec_2){
    double point;
    int i;
    point = 0.0;
    for(i=0;i<3;i++){
        point+=vec_1[i]*vec_2[i];
    }
    return point;
}


__device__ void cross_product(FLOAT *vec_1, FLOAT *vec_2, FLOAT *result){
    result[0] = vec_1[1]*vec_2[2] - vec_1[2]*vec_2[1];
    result[1] = vec_1[2]*vec_2[0] - vec_1[0]*vec_2[2];
    result[2] = vec_1[0]*vec_2[1] - vec_1[1]*vec_2[0];
    return;
}


__device__ void normalize(FLOAT *vec){
    FLOAT norm;
    int i;
    norm = 0.0;
    norm = point_product(vec, vec);
    norm = sqrt(norm);
    for(i=0;i<3;i++){
        vec[i] = vec[i]/norm;
    }
    return;
}
